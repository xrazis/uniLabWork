#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime_api.h>
#include "book.h"

// Problem size
#define NI 4096
#define NJ 4096
#define blockSize 32

__global__ void convolution(const double *A, double *B) {
    int i, j;
    double c11, c12, c13, c21, c22, c23, c31, c32, c33;

    c11 = +0.2;
    c21 = +0.5;
    c31 = -0.8;
    c12 = -0.3;
    c22 = +0.6;
    c32 = -0.9;
    c13 = +0.4;
    c23 = +0.7;
    c33 = +0.10;

    // Idiomatic CUDA
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int stepRow = blockDim.x * gridDim.x;
    int stepCol = blockDim.y * gridDim.y;

    for (i = row; i < NI; i += stepRow) {
        for (j = col; j < NJ; j += stepCol) {
            B[i * NJ + j] =
                    c11 * A[(i - 1) * NJ + (j - 1)] + c12 * A[(i + 0) * NJ + (j - 1)] +
                    c13 * A[(i + 1) * NJ + (j - 1)] + c21 * A[(i - 1) * NJ + (j + 0)] +
                    c22 * A[(i + 0) * NJ + (j + 0)] + c23 * A[(i + 1) * NJ + (j + 0)] +
                    c31 * A[(i - 1) * NJ + (j + 1)] + c32 * A[(i + 0) * NJ + (j + 1)] +
                    c33 * A[(i + 1) * NJ + (j + 1)];
        }
    }
}

// Definitely not the fastest way. % operator is very slow and highly divergent
// warps are very inefficient (see https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf)
__global__ void findMaxInDiagonal(double *B, double *max) {
    // Idiomatic CUDA
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int c_t = threadIdx.x;
    int r_t = threadIdx.y;

    // 2D -> 1D index
    int pos_1D = row * NJ + col;
    int pos_1D_t = r_t * blockDim.x + c_t;

    // Shared memory
    extern __shared__ double shared[];
    shared[pos_1D_t] = B[pos_1D];

    // Do reduction in shared memory
    __syncthreads();
    for (int s = (blockDim.x * blockDim.y) / 2; s > 0; s >>= 1) {
        if (pos_1D_t < s)
            shared[pos_1D_t] = fmax(shared[pos_1D_t], shared[pos_1D_t + s]);
        __syncthreads();
    }

    if (r_t == 0 && c_t == 0)
        max[blockIdx.y * gridDim.x + blockIdx.x] = shared[0];
}

void init(double *A) {
    int i, j;

    for (i = 0; i < NI; ++i) {
        for (j = 0; j < NJ; ++j) {
            A[i * NJ + j] = (double) rand() / RAND_MAX;
        }
    }
}

int main() {
    double *local_A, *A, *local_B, *B, *local_max, *max;
    struct timeval cpu_start{}, cpu_end{};
    int size = NI * NJ * sizeof(double);

    gettimeofday(&cpu_start, nullptr);

    // Allocate CPU memory
    local_A = (double *) malloc(size);
    local_B = (double *) malloc(size);
    local_max = (double *) malloc(sizeof(double));

    // Allocate GPU memory - Unified memory won't work with older GPU architectures, see requirements ->
    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#um-requirements
    HANDLE_ERROR(hipMalloc((void **) &A, size));
    HANDLE_ERROR(hipMalloc((void **) &B, size));
    HANDLE_ERROR(hipMalloc((void **) &max, sizeof(double)));

    // Initialize local_A on the host
    init(local_A);

    // Copy memory from host to device
    HANDLE_ERROR(hipMemcpy(A, local_A, size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(B, local_B, size, hipMemcpyHostToDevice));

    // Setup execution parameters
    dim3 threads(blockSize, blockSize);
    dim3 grid(NI / threads.x, NJ / threads.y);

    // Invoke the kernel
    convolution<<<1, threads>>>(A, B);

    // Copy memory from host to device
    HANDLE_ERROR(hipMemcpy(max, local_max, sizeof(double), hipMemcpyHostToDevice));

    // Invoke the kernel
    findMaxInDiagonal<<<1, 2, blockSize * blockSize * sizeof(double)>>>(B, max);

    // Copy max back from the GPU to the CPU
    HANDLE_ERROR(hipMemcpy(local_B, B, size, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(local_max, max, sizeof(double), hipMemcpyDeviceToHost));

    hipDeviceSynchronize();

    gettimeofday(&cpu_end, nullptr);

    fprintf(stdout, "CPU Runtime: %0.6lfs\n",
            ((cpu_end.tv_sec - cpu_start.tv_sec) * 1000000.0 + (cpu_end.tv_usec - cpu_start.tv_usec)) / 1000000.0);

    printf("Max:\t%f\t%f\n", *local_max, local_B[0]);

    // Free memory
    free(local_A);
    free(local_B);
    free(local_max);
    hipFree(A);
    hipFree(B);
    hipFree(max);

    return 0;
}

// /usr/local/cuda-11.5/bin/nvcc -arch=compute_50 -code=sm_50 ./main.cu -o main && /usr/local/cuda-11.5/bin/nvprof ./main